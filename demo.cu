#include "hip/hip_runtime.h"
// Author : Przemyslaw Zaworski
// Compile: nvcc -o demo.exe demo.cu -lopengl32 -arch=sm_30  user32.lib gdi32.lib
// Usage pattern: demo filename mapsize steps cminr cming cminb cmaxr cmaxg cmaxb intensity threshold crangemin crangemax screenwidth screenheight mode voxelfile offx offy offz
// Example usage: demo obraz3LW_A.vtk 128 128 0.0 0.0 1.0 0.1 0.0 0.0 1.0 1.0 0.0 1.0 1280.0 720.0 0 image.bin 0.25 0.25 0.0

#include <windows.h>
#include <GL/gl.h>
#include <stddef.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <shellapi.h>

// Screen settings
#define FieldOfView 60.0f
#define NearClip 0.01f
#define FarClip 1000.0f
#define VerticalSync 0

// Load selected Win32 API & OpenGL functions
typedef GLuint(WINAPI *PFNGLCREATEPROGRAMPROC) ();
typedef GLuint(WINAPI *PFNGLCREATESHADERPROC) (GLenum t);
typedef void(WINAPI *PFNGLSHADERSOURCEPROC) (GLuint s, GLsizei c, const char*const*string, const GLint* i);
typedef void(WINAPI *PFNGLCOMPILESHADERPROC) (GLuint s);
typedef void(WINAPI *PFNGLATTACHSHADERPROC) (GLuint p, GLuint s);
typedef void(WINAPI *PFNGLLINKPROGRAMPROC) (GLuint p);
typedef void(WINAPI *PFNGLUSEPROGRAMPROC) (GLuint p);
typedef void(WINAPI *PFNGLGENBUFFERSPROC) (GLsizei n, GLuint *b);
typedef void(WINAPI *PFNGLBINDBUFFERPROC) (GLenum t, GLuint b);
typedef void(WINAPI *PFNGLBUFFERDATAPROC) (GLenum t, ptrdiff_t s, const GLvoid *d, GLenum u);
typedef void(WINAPI *PFNGLBINDVERTEXARRAYPROC) (GLuint a);
typedef void(WINAPI *PFNGLENABLEVERTEXATTRIBARRAYPROC) (GLuint i);
typedef void(WINAPI *PFNGLVERTEXATTRIBPOINTERPROC) (GLuint i, GLint s, GLenum t, GLboolean n, GLsizei k, const void *p);
typedef void(WINAPI *PFNGLDISABLEVERTEXATTRIBARRAYPROC) (GLuint i);
typedef int(WINAPI *PFNWGLSWAPINTERVALEXTPROC) (int i);
typedef int(WINAPI *PFNGLGETUNIFORMLOCATIONPROC) (GLuint p, const char *n);
typedef void(WINAPI *PFNGLGENVERTEXARRAYSPROC) (GLsizei n, GLuint *a);
typedef void(WINAPI *PFNGLUNIFORMMATRIX4FVPROC) (GLint l, GLsizei c, GLboolean t, const GLfloat *v);
typedef void(WINAPI *PFNGLUNIFORM1IPROC) (GLint l, GLint v);
typedef void(WINAPI *PFNGLACTIVETEXTUREPROC) (GLenum t);
typedef void(WINAPI *PFNGLUNIFORM3FPROC) (GLint location, float v0, float v1, float v2);
typedef void(WINAPI *PFNGLGETSHADERIVPROC) (GLuint s, GLenum v, GLint *p);
typedef void(WINAPI *PFNGLGETSHADERINFOLOGPROC) (GLuint s, GLsizei b, GLsizei *l, char *i);
typedef void(WINAPI *PFNGLUNIFORM1FPROC) (GLint l, GLfloat v0);
typedef void(WINAPI *PFNGLTEXIMAGE3DPROC) (GLenum a, GLint l, GLint i, GLsizei w, GLsizei h, GLsizei d, GLint b, GLenum f, GLenum t, const void *p);

PFNGLCREATEPROGRAMPROC glCreateProgram;
PFNGLCREATESHADERPROC glCreateShader;
PFNGLSHADERSOURCEPROC glShaderSource;
PFNGLCOMPILESHADERPROC glCompileShader;
PFNGLATTACHSHADERPROC glAttachShader;
PFNGLLINKPROGRAMPROC glLinkProgram;
PFNGLUSEPROGRAMPROC glUseProgram;
PFNGLGENBUFFERSPROC glGenBuffers;
PFNGLBINDBUFFERPROC glBindBuffer;
PFNGLBUFFERDATAPROC glBufferData;
PFNGLBINDVERTEXARRAYPROC glBindVertexArray;
PFNGLENABLEVERTEXATTRIBARRAYPROC glEnableVertexAttribArray;
PFNGLVERTEXATTRIBPOINTERPROC glVertexAttribPointer;
PFNGLDISABLEVERTEXATTRIBARRAYPROC glDisableVertexAttribArray;
PFNWGLSWAPINTERVALEXTPROC wglSwapIntervalEXT;
PFNGLGETUNIFORMLOCATIONPROC glGetUniformLocation;
PFNGLGENVERTEXARRAYSPROC glGenVertexArrays;
PFNGLUNIFORMMATRIX4FVPROC glUniformMatrix4fv;
PFNGLUNIFORM1IPROC glUniform1i;
PFNGLACTIVETEXTUREPROC glActiveTexture;
PFNGLUNIFORM3FPROC glUniform3f;
PFNGLGETSHADERIVPROC glGetShaderiv;
PFNGLGETSHADERINFOLOGPROC glGetShaderInfoLog;
PFNGLUNIFORM1FPROC glUniform1f;
PFNGLTEXIMAGE3DPROC glTexImage3D;

void glInit()
{
	glCreateProgram = (PFNGLCREATEPROGRAMPROC)wglGetProcAddress("glCreateProgram");
	glCreateShader = (PFNGLCREATESHADERPROC)wglGetProcAddress("glCreateShader");
	glShaderSource = (PFNGLSHADERSOURCEPROC)wglGetProcAddress("glShaderSource");
	glCompileShader = (PFNGLCOMPILESHADERPROC)wglGetProcAddress("glCompileShader");
	glAttachShader = (PFNGLATTACHSHADERPROC)wglGetProcAddress("glAttachShader");
	glLinkProgram = (PFNGLLINKPROGRAMPROC)wglGetProcAddress("glLinkProgram");
	glUseProgram = (PFNGLUSEPROGRAMPROC)wglGetProcAddress("glUseProgram");
	glGenBuffers = (PFNGLGENBUFFERSPROC)wglGetProcAddress("glGenBuffers");
	glBindBuffer = (PFNGLBINDBUFFERPROC)wglGetProcAddress("glBindBuffer");
	glBufferData = (PFNGLBUFFERDATAPROC)wglGetProcAddress("glBufferData");
	glBindVertexArray = (PFNGLBINDVERTEXARRAYPROC)wglGetProcAddress("glBindVertexArray");
	glEnableVertexAttribArray = (PFNGLENABLEVERTEXATTRIBARRAYPROC)wglGetProcAddress("glEnableVertexAttribArray");
	glVertexAttribPointer = (PFNGLVERTEXATTRIBPOINTERPROC)wglGetProcAddress("glVertexAttribPointer");
	glDisableVertexAttribArray = (PFNGLDISABLEVERTEXATTRIBARRAYPROC)wglGetProcAddress("glDisableVertexAttribArray");
	wglSwapIntervalEXT = (PFNWGLSWAPINTERVALEXTPROC)wglGetProcAddress("wglSwapIntervalEXT");
	glGetUniformLocation = (PFNGLGETUNIFORMLOCATIONPROC)wglGetProcAddress("glGetUniformLocation");
	glGenVertexArrays = (PFNGLGENVERTEXARRAYSPROC)wglGetProcAddress("glGenVertexArrays");
	glUniformMatrix4fv = (PFNGLUNIFORMMATRIX4FVPROC)wglGetProcAddress("glUniformMatrix4fv");
	glUniform1i = (PFNGLUNIFORM1IPROC)wglGetProcAddress("glUniform1i");
	glActiveTexture = (PFNGLACTIVETEXTUREPROC)wglGetProcAddress("glActiveTexture");
	glUniform3f = (PFNGLUNIFORM3FPROC)wglGetProcAddress("glUniform3f");
	glGetShaderiv = (PFNGLGETSHADERIVPROC)wglGetProcAddress("glGetShaderiv");
	glGetShaderInfoLog = (PFNGLGETSHADERINFOLOGPROC)wglGetProcAddress("glGetShaderInfoLog");
	glUniform1f = (PFNGLUNIFORM1FPROC)wglGetProcAddress("glUniform1f");
	glTexImage3D = (PFNGLTEXIMAGE3DPROC) wglGetProcAddress("glTexImage3D");
}

// Global variables
unsigned int VertexBuffer, VertexArrayID;
unsigned char* device;
float* input;
float* output;
float* inputColors;
float* outputColors;	
float offsetX = 0.0f, offsetZ = 0.0f;
float CameraRotYX[4][4], CameraRotYXZ[4][4]; 
float CameraTR[4][4], CameraMatrix[4][4], ViewMatrix[4][4];
float ProjectionViewMatrix[4][4], MVP[4][4];
int cells = 0;
int points = 0;
float sminx = 0.01f;
float smaxx = 0.99f;
float sminy = 0.01f;
float smaxy = 0.99f;
float sminz = 0.01f;
float smaxz = 0.99f;
bool DebugImage = false;

struct DataSet 
{
	float* vertices;
	float* scalars;
};

// 3D coordinates of cube in object space
static const GLfloat vertices[] = 
{
	-0.5f,-0.5f,-0.5f,
	-0.5f,-0.5f, 0.5f,
	-0.5f, 0.5f, 0.5f, 
	 0.5f, 0.5f,-0.5f, 
	-0.5f,-0.5f,-0.5f,
	-0.5f, 0.5f,-0.5f, 
	 0.5f,-0.5f, 0.5f,
	-0.5f,-0.5f,-0.5f,
	 0.5f,-0.5f,-0.5f,
	 0.5f, 0.5f,-0.5f,
	 0.5f,-0.5f,-0.5f,
	-0.5f,-0.5f,-0.5f,
	-0.5f,-0.5f,-0.5f,
	-0.5f, 0.5f, 0.5f,
	-0.5f, 0.5f,-0.5f,
	 0.5f,-0.5f, 0.5f,
	-0.5f,-0.5f, 0.5f,
	-0.5f,-0.5f,-0.5f,
	-0.5f, 0.5f, 0.5f,
	-0.5f,-0.5f, 0.5f,
	 0.5f,-0.5f, 0.5f,
	 0.5f, 0.5f, 0.5f,
	 0.5f,-0.5f,-0.5f,
	 0.5f, 0.5f,-0.5f,
	 0.5f,-0.5f,-0.5f,
	 0.5f, 0.5f, 0.5f,
	 0.5f,-0.5f, 0.5f,
	 0.5f, 0.5f, 0.5f,
	 0.5f, 0.5f,-0.5f,
	-0.5f, 0.5f,-0.5f,
	 0.5f, 0.5f, 0.5f,
	-0.5f, 0.5f,-0.5f,
	-0.5f, 0.5f, 0.5f,
	 0.5f, 0.5f, 0.5f,
	-0.5f, 0.5f, 0.5f,
	 0.5f,-0.5f, 0.5f
};

// Remap value x in range(a,b) to range (c,d)
float remap (float x, float a, float b, float c, float d)  
{
	return (x-a)/(b-a)*(d-c) + c; 
}

// Convert degrees to radians
float deg2rad(float x) 
{
	return (x * 3.14159265358979323846f / 180.0f);
}

// 4x4 matrices multiplication
void Mul(float mat1[][4], float mat2[][4], float res[][4])
{
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			res[i][j] = 0;
			for (int k = 0; k < 4; k++) 
			{
				res[i][j] += mat1[i][k]*mat2[k][j];
			}
		}
	}
}

// Matrix 4x4 inversion
void Inverse( float param[4][4], float k[4][4])
{
	float invOut[16];
	float m[16] = 
	{
		param[0][0],param[0][1],param[0][2],param[0][3],
		param[1][0],param[1][1],param[1][2],param[1][3],
		param[2][0],param[2][1],param[2][2],param[2][3],
		param[3][0],param[3][1],param[3][2],param[3][3]
	};
	float inv[16], det;
	int i;
	inv[0]  =  m[5]*m[10]*m[15]-m[5]*m[11]*m[14]-m[9]*m[6]*m[15]+m[9]*m[7]*m[14]+m[13]*m[6]*m[11]-m[13]*m[7]*m[10];
	inv[4]  = -m[4]*m[10]*m[15]+m[4]*m[11]*m[14]+m[8]*m[6]*m[15]-m[8]*m[7]*m[14]-m[12]*m[6]*m[11]+m[12]*m[7]*m[10];
	inv[8]  =  m[4] *m[9]*m[15]-m[4]*m[11]*m[13]-m[8]*m[5]*m[15]+m[8]*m[7]*m[13]+m[12]*m[5]*m[11]-m[12]*m[7] *m[9];
	inv[12] = -m[4] *m[9]*m[14]+m[4]*m[10]*m[13]+m[8]*m[5]*m[14]-m[8]*m[6]*m[13]-m[12]*m[5]*m[10]+m[12]*m[6] *m[9];
	inv[1]  = -m[1]*m[10]*m[15]+m[1]*m[11]*m[14]+m[9]*m[2]*m[15]-m[9]*m[3]*m[14]-m[13]*m[2]*m[11]+m[13]*m[3]*m[10];
	inv[5]  =  m[0]*m[10]*m[15]-m[0]*m[11]*m[14]-m[8]*m[2]*m[15]+m[8]*m[3]*m[14]+m[12]*m[2]*m[11]-m[12]*m[3]*m[10];
	inv[9]  = -m[0] *m[9]*m[15]+m[0]*m[11]*m[13]+m[8]*m[1]*m[15]-m[8]*m[3]*m[13]-m[12]*m[1]*m[11]+m[12]*m[3] *m[9];
	inv[13] =  m[0] *m[9]*m[14]-m[0]*m[10]*m[13]-m[8]*m[1]*m[14]+m[8]*m[2]*m[13]+m[12]*m[1]*m[10]-m[12]*m[2] *m[9];
	inv[2]  =  m[1] *m[6]*m[15]-m[1] *m[7]*m[14]-m[5]*m[2]*m[15]+m[5]*m[3]*m[14]+m[13]*m[2] *m[7]-m[13]*m[3] *m[6];
	inv[6]  = -m[0] *m[6]*m[15]+m[0] *m[7]*m[14]+m[4]*m[2]*m[15]-m[4]*m[3]*m[14]-m[12]*m[2] *m[7]+m[12]*m[3] *m[6];
	inv[10] =  m[0] *m[5]*m[15]-m[0] *m[7]*m[13]-m[4]*m[1]*m[15]+m[4]*m[3]*m[13]+m[12]*m[1] *m[7]-m[12]*m[3] *m[5];
	inv[14] = -m[0] *m[5]*m[14]+m[0] *m[6]*m[13]+m[4]*m[1]*m[14]-m[4]*m[2]*m[13]-m[12]*m[1] *m[6]+m[12]*m[2] *m[5];
	inv[3]  = -m[1] *m[6]*m[11]+m[1] *m[7]*m[10]+m[5]*m[2]*m[11]-m[5]*m[3]*m[10] -m[9]*m[2] *m[7] +m[9]*m[3] *m[6];
	inv[7]  =  m[0] *m[6]*m[11]-m[0] *m[7]*m[10]-m[4]*m[2]*m[11]+m[4]*m[3]*m[10] +m[8]*m[2] *m[7] -m[8]*m[3] *m[6];
	inv[11] = -m[0] *m[5]*m[11]+m[0] *m[7]*m[9] +m[4]*m[1]*m[11]-m[4]*m[3] *m[9] -m[8]*m[1] *m[7] +m[8]*m[3] *m[5];
	inv[15] =  m[0] *m[5]*m[10]-m[0] *m[6]*m[9] -m[4]*m[1]*m[10]+m[4]*m[2] *m[9] +m[8]*m[1] *m[6] -m[8]*m[2] *m[5];
	det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];
	det = 1.0 / det;
	for (i = 0; i < 16; i++) invOut[i] = inv[i] * det;	
	k[0][0] = invOut[0];  k[0][1] = invOut[1];  k[0][2] = invOut[2];  k[0][3] = invOut[3];
	k[1][0] = invOut[4];  k[1][1] = invOut[5];  k[1][2] = invOut[6];  k[1][3] = invOut[7];
	k[2][0] = invOut[8];  k[2][1] = invOut[9];  k[2][2] = invOut[10]; k[2][3] = invOut[11];
	k[3][0] = invOut[12]; k[3][1] = invOut[13]; k[3][2] = invOut[14]; k[3][3] = invOut[15];  
}

float clamp(float x, float a, float b)
{
	return fmaxf(a, fminf(b, x));
}

// GLSL vertex shader code
static const char* VertexShader = \
	"#version 430 core\n"
	"layout (location=0) in vec3 vertexPosition;"	
	"out vec3 world;"
	"uniform mat4 MVP;"
	"void main()"
	"{"	
		"gl_Position = MVP * vec4(vertexPosition,1.0);"
		"world = vertexPosition;"
	"}";

// GLSL fragment(pixel) shader code	
static const char* FragmentShader = \
	"#version 430 core\n"
	"out vec4 color;"
	"in vec3 world;"
	"uniform int steps;"
	"uniform sampler3D _MainTex;"	
	"uniform vec3 _WorldSpaceCameraPos;"
	"uniform float _Intensity;"
	"uniform float _Threshold;"
	"uniform vec3 _SliceMin ;"
	"uniform vec3 _SliceMax ;"
	"uniform vec3 _ColorMin;"
	"uniform vec3 _ColorMax;"	
	"mat4 _AxisRotationMatrix = mat4(1.0,0.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,0.0,1.0,0.0,0.0,0.0,0.0,1.0);"
	"float SampleVolume(vec3 uv, vec3 p)"
	"{"
		"vec3 axis = (   _AxisRotationMatrix * vec4(p, 0) ).xyz + 0.5;"
		"float min = step(_SliceMin.x, axis.x) * step(_SliceMin.y, axis.y) * step(_SliceMin.z, axis.z);"
		"float max = step(axis.x, _SliceMax.x) * step(axis.y, _SliceMax.y) * step(axis.z, _SliceMax.z);"
		"return texture(_MainTex, uv).r * _Intensity * min * max;"
	"}"
	"void main()"
	"{"	
		"vec3 ro = world;"
		"vec3 rd = normalize(world - _WorldSpaceCameraPos);	"		 
		"vec3 AABBmin = vec3(-0.5, -0.5, -0.5);"
		"vec3 AABBmax = vec3(0.5, 0.5, 0.5);"
		"vec3 tbot = (1.0 / rd) * (AABBmin - ro);"
		"vec3 ttop = (1.0 / rd) * (AABBmax - ro);"
		"vec3 tmin = min(ttop, tbot);"
		"vec3 tmax = max(ttop, tbot);"
		"vec2 a = max(tmin.xx, tmin.yz);"
		"float tnear = max(0.0,max(a.x, a.y));"
		"vec2 b = min(tmax.xx, tmax.yz);"
		"float tfar = min(b.x, b.y);"
		"vec3 end = ro + rd * tfar;"
		"vec3 d = normalize(end - ro) * (abs(tfar - tnear) / float(steps));"
		"vec4 t = vec4(0, 0, 0, 0);"
		"for (int i = 0; i < steps; i++)"
		"{"
			"float v = SampleVolume(ro+0.5, ro);"
			"vec4 s = vec4(v, v, v, v);"
			"s.a *= 0.5;"
			"s.rgb *= s.a;"
			"t = (1.0 - t.a) * s + t;"
			"ro += d;"
			"if (t.a > _Threshold) break;"
		"}"
		"color =  clamp(t, 0.0,1.0).rgba;"
		"if (t.r>0.005) "
		"color = vec4(mix(_ColorMin,_ColorMax,t.r),t.a);"
		"else "
		"color = vec4(0.0,0.0,0.0,t.a);"
	"}";

// Compute determinant of 4x4 matrix on the GPU
__device__ float Determinant (float m[4][4]) 
{
	return
	m[0][3] * m[1][2] * m[2][1] * m[3][0] - m[0][2] * m[1][3] * m[2][1] * m[3][0] -
	m[0][3] * m[1][1] * m[2][2] * m[3][0] + m[0][1] * m[1][3] * m[2][2] * m[3][0] +
	m[0][2] * m[1][1] * m[2][3] * m[3][0] - m[0][1] * m[1][2] * m[2][3] * m[3][0] -
	m[0][3] * m[1][2] * m[2][0] * m[3][1] + m[0][2] * m[1][3] * m[2][0] * m[3][1] +
	m[0][3] * m[1][0] * m[2][2] * m[3][1] - m[0][0] * m[1][3] * m[2][2] * m[3][1] -
	m[0][2] * m[1][0] * m[2][3] * m[3][1] + m[0][0] * m[1][2] * m[2][3] * m[3][1] +
	m[0][3] * m[1][1] * m[2][0] * m[3][2] - m[0][1] * m[1][3] * m[2][0] * m[3][2] -
	m[0][3] * m[1][0] * m[2][1] * m[3][2] + m[0][0] * m[1][3] * m[2][1] * m[3][2] +
	m[0][1] * m[1][0] * m[2][3] * m[3][2] - m[0][0] * m[1][1] * m[2][3] * m[3][2] -
	m[0][2] * m[1][1] * m[2][0] * m[3][3] + m[0][1] * m[1][2] * m[2][0] * m[3][3] +
	m[0][2] * m[1][0] * m[2][1] * m[3][3] - m[0][0] * m[1][2] * m[2][1] * m[3][3] -
	m[0][1] * m[1][0] * m[2][2] * m[3][3] + m[0][0] * m[1][1] * m[2][2] * m[3][3];
}

// Compute sign on the GPU
__device__ float Sign (float x)
{
	return ( (0.0f < x) - (x < 0.0f) );
}

// Compute on the GPU whether given point is inside tetrahedron
__device__ bool InsideTetrahedronLegacy (float v1[3], float v2[3], float v3[3], float v4[3], float p[3])
{
	float D0[4][4] = 
	{
		{v1[0],v1[1],v1[2],1.0f},{v2[0],v2[1],v2[2],1.0f},{v3[0],v3[1],v3[2],1.0f},{v4[0],v4[1],v4[2],1.0f}
	};
	float D1[4][4] = 
	{
		{p[0],p[1],p[2],1.0f},{v2[0],v2[1],v2[2],1.0f},{v3[0],v3[1],v3[2],1.0f},{v4[0],v4[1],v4[2],1.0f}
	};
	float D2[4][4] = 
	{
		{v1[0],v1[1],v1[2],1.0f},{p[0],p[1],p[2],1.0f},{v3[0],v3[1],v3[2],1.0f},{v4[0],v4[1],v4[2],1.0f}
	};
	float D3[4][4] = 
	{
		{v1[0],v1[1],v1[2],1.0f},{v2[0],v2[1],v2[2],1.0f},{p[0],p[1],p[2],1.0f},{v4[0],v4[1],v4[2],1.0f}
	};
	float D4[4][4] = 
	{
		{v1[0],v1[1],v1[2],1.0f},{v2[0],v2[1],v2[2],1.0f},{v3[0],v3[1],v3[2],1.0f},{p[0],p[1],p[2],1.0f}
	};
	float a = Determinant(D0);
	float b = Determinant(D1);
	float c = Determinant(D2);
	float d = Determinant(D3);
	float e = Determinant(D4);
	return ( (Sign(a)==Sign(b)) &&  (Sign(a)==Sign(c)) && (Sign(a)==Sign(d)) && (Sign(a)==Sign(e)) );
}

// Compute on the GPU whether given point is inside tetrahedron - faster version	
__device__ bool InsideTetrahedron (float a[3], float b[3], float c[3], float d[3], float p[3])
{
	float vap[3] = {p[0] - a[0], p[1] - a[1], p[2] - a[2]};
	float vbp[3] = {p[0] - b[0], p[1] - b[1], p[2] - b[2]};
	float vab[3] = {b[0] - a[0], b[1] - a[1], b[2] - a[2]};
	float vac[3] = {c[0] - a[0], c[1] - a[1], c[2] - a[2]};
	float vad[3] = {d[0] - a[0], d[1] - a[1], d[2] - a[2]};
	float vbc[3] = {c[0] - b[0], c[1] - b[1], c[2] - b[2]};
	float vbd[3] = {d[0] - b[0], d[1] - b[1], d[2] - b[2]};
	float x[3] = {vbd[1]*vbc[2]-vbd[2]*vbc[1], vbd[2]*vbc[0]-vbd[0]*vbc[2], vbd[0]*vbc[1]-vbd[1]*vbc[0]};
	float y[3] = {vac[1]*vad[2]-vac[2]*vad[1], vac[2]*vad[0]-vac[0]*vad[2], vac[0]*vad[1]-vac[1]*vad[0]};
	float z[3] = {vad[1]*vab[2]-vad[2]*vab[1], vad[2]*vab[0]-vad[0]*vab[2], vad[0]*vab[1]-vad[1]*vab[0]};
	float w[3] = {vab[1]*vac[2]-vab[2]*vac[1], vab[2]*vac[0]-vab[0]*vac[2], vab[0]*vac[1]-vab[1]*vac[0]};		
	float va6 = vbp[0] * x[0] + vbp[1] * x[1] + vbp[2] * x[2];     
	float vb6 = vap[0] * y[0] + vap[1] * y[1] + vap[2] * y[2];   
	float vc6 = vap[0] * z[0] + vap[1] * z[1] + vap[2] * z[2];
	float vd6 = vap[0] * w[0] + vap[1] * w[1] + vap[2] * w[2];
	float q[3] = {vac[1]*vad[2]-vac[2]*vad[1], vac[2]*vad[0]-vac[0]*vad[2], vac[0]*vad[1]-vac[1]*vad[0]};
	float v6 = 1.0f / (vab[0] * q[0] + vab[1] * q[1] + vab[2] * q[2]);
	float k[4] =  {va6*v6, vb6*v6, vc6*v6, vd6*v6};
	return ((k[0] >= 0.0) && (k[0] <= 1.0) && (k[1] >= 0.0) && (k[1] <= 1.0) && (k[2] >= 0.0) && (k[2] <= 1.0) && (k[3] >= 0.0) && (k[3] <= 1.0)) ? true : false;
}

// CUDA kernel, to generate voxel map
__global__ void GenerateVoxelMap (unsigned char* color, float* v, float* s, int k, int msize) 
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
	unsigned int i = x + y * msize + z * msize * msize ;
	if (color[i]>0) return;    
	float3 resolution = make_float3(msize,msize,msize);
	float3 coordinates = make_float3((float)x, (float)y,float(z));
	float3 uv = make_float3 (coordinates.x / resolution.x, coordinates.y / resolution.y, coordinates.z / resolution.z );
	float a[3] = {v[k*12+0], v[k*12+1], v[k*12+2]};
	float b[3] = {v[k*12+3], v[k*12+4], v[k*12+5]};
	float c[3] = {v[k*12+6], v[k*12+7], v[k*12+8]};
	float d[3] = {v[k*12+9], v[k*12+10], v[k*12+11]};
	float p[3] = {uv.x,uv.y,uv.z};
	if (InsideTetrahedron(a, b, c, d, p))
	{
		float f = (s[k*4+0] + s[k*4+1] + s[k*4+2] + s[k*4+3]) / 4.0f;
		color[i] = (unsigned char)(f*255);
	}
	else 
	{
		color[i] = 0;
	}
}

// Declare matrices for building ModelViewProjection matrix.
float ModelMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,1.0f,0.0f,
	0.0f,0.0f,0.0f,1.0f
};

float CameraTranslationMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,1.0f,-5.0f,
	0.0f,0.0f,0.0f,1.0f
};

float CameraRotationYMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,1.0f,0.0f,
	0.0f,0.0f,0.0f,1.0f
};

float CameraRotationXMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,1.0f,0.0f,
	0.0f,0.0f,0.0f,1.0f
};

float CameraRotationZMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,1.0f,0.0f,
	0.0f,0.0f,0.0f,1.0f
};

float CameraScaleMatrix[4][4] = 
{
	1.0f,0.0f,0.0f,0.0f,
	0.0f,1.0f,0.0f,0.0f,
	0.0f,0.0f,-1.0f,0.0f,
	0.0f,0.0f,0.0f,1.0f
};

float ProjectionMatrix[4][4] = 
{
	0.0f,0.0f,0.0f,0.0f,
	0.0f,0.0f,0.0f,0.0f,
	0.0f,0.0f,0.0f,0.0f,
	0.0f,0.0f,-1.0f,0.0f
};

// Shader debugging
void Debug(int sh)
{
	GLint isCompiled = 0;
	glGetShaderiv(sh,0x8B82,&isCompiled);
	if(isCompiled == GL_FALSE)
	{
		GLint length = 0;
		glGetShaderiv(sh,0x8B84,&length);
		GLsizei q = 0;
		char* log = (char*)malloc(sizeof(char)*length);
		glGetShaderInfoLog(sh,length,&q,log);
		if (length>1)
		{
			FILE *file = fopen ("debug.log","a");
			fprintf (file,"%s\n%s\n",(char*)glGetString(0x8B8C),log);
			fclose (file);
			ExitProcess(0);
		}
		free(log);
	}
}

// Compile shaders	
int MakeShader(const char* VS, const char* FS)
{
	int p = glCreateProgram();
	int s1 = glCreateShader(0x8B31);
	int s2 = glCreateShader(0x8B30);
	glShaderSource(s1,1,&VS,0);
	glShaderSource(s2,1,&FS,0);	
	glCompileShader(s1);
	glCompileShader(s2);
	glAttachShader(p,s1);
	glAttachShader(p,s2);
	glLinkProgram(p);
	Debug(s2);
	return p;
}

// Load data from VTK file.
struct DataSet StreamData (const char* path, float ColorRangeMin, float ColorRangeMax, float ox, float oy, float oz)
{
	struct DataSet grid; 
	FILE* fp = fopen(path, "r");
	char line[200];
	int v=0;
	int m=0;
	int o=0;
	int i=0;
	char *token;
	int state = 0;
	float MinX = 100.0f;
	float MaxX = -100.0f;
	float MinY = 100.0f;
	float MaxY = -100.0f;
	float MinZ = 100.0f;
	float MaxZ = -100.0f;
	float CMin = 100.0f;
	float CMax = -100.0f;
	
	while (1) //read number of cells
	{
		if (fgets(line,150, fp) == NULL) break;
		if(strcmp(line, "\n") == 0) 
		{
			state++;
			continue;
		}
		i++;
		if (i<5) continue;
		if (state==0)
		{
			continue;
		}
		if (state==1)
		{
			token = strtok(line, " ");
			if(strcmp(token, "CELLS") == 0) 
			{
				token = strtok(NULL," ");
				int hh = atoll(token);
				cells = hh;
				points = 4 * hh;
				break;
			}		
		}	
		if (state==2)
		{
			continue;
		}
		if (state==3)
		{
			continue;
		}
	} 

	printf( "Number of cells: %d\n", cells );
	fseek(fp, 0, 0)	;
	v=m=o=i=state=0;
	int p=0;
	
	input = (float*)malloc(4*3*points);
	output = (float*)malloc(4*3*points);
	inputColors = (float*)malloc(4*points);
	outputColors = (float*)malloc(4*points);
	
	while (1) //read vertices and calculate bounds
	{
		if (fgets(line,150, fp) == NULL) break;
		if(strcmp(line, "\n") == 0) 
		{
			state++;
			continue;
		}
		i++;
		if (i<5) continue;
		if (state==0)
		{
			token = strtok(line, " ");
			if(strcmp(token, "POINTS") == 0) continue;
			while( token != NULL ) 
			{
				float h = atof(token);
				input[v] = h;
				v++;
				token = strtok(NULL," ");
			}
		}
		
		if (state==1)
		{
			token = strtok(line, " ");
			if(strcmp(token, "CELLS") == 0) 
			{	
				token = strtok(NULL," ");
				int hh = atoll(token);
				cells = hh;
				points = 4 * hh;
				continue;
			}
			p=0;
			while( token != NULL ) 
			{
				if (p==0) 
				{
					p++;
					token = strtok(NULL," ");
					continue;
				}
				int h = atoll(token);
				output[3*m] = input[3*h];
				output[3*m+1] = input[3*h+1];
				output[3*m+2] = input[3*h+2];
				if (input[3*m]<MinX) MinX = input[3*m];
				if (input[3*m]>MaxX) MaxX = input[3*m];
				if (input[3*m+1]<MinY) MinY = input[3*m+1];
				if (input[3*m+1]>MaxY) MaxY = input[3*m+1];
				if (input[3*m+2]<MinZ) MinZ = input[3*m+2];
				if (input[3*m+2]>MaxZ) MaxZ = input[3*m+2];
				m++;
				token = strtok(NULL," ");
			}		
		}		
		if (state==2)
		{
			continue;
		}
		if (state==3)
		{
			token = strtok(line, " ");
			if(strcmp(token, "POINT_DATA") == 0) continue;
			if(strcmp(token, "SCALARS") == 0) continue;
			if(strcmp(token, "LOOKUP_TABLE") == 0) continue;
			while( token != NULL ) 
			{
				float h = atof(token);
				inputColors[o] = h;
				if (h<CMin) CMin = h;
				if (h>CMax) CMax = h;
				o++;
				token = strtok(NULL," ");
			}
		}
	} 

	printf( "Scalars range: %f %f [remapped to range %f %f]\n", CMin, CMax, ColorRangeMin, ColorRangeMax );
	fseek(fp, 0, 0)	;	
	v=m=o=i=state=p=0;

	float deltaA = abs(MaxX - MinX);
	float deltaB = abs(MaxY - MinY);
	float deltaC = abs(MaxZ - MinZ);
	float MaxDelta = max(deltaA,max(deltaB,deltaC));
	float ScaleX = deltaA/MaxDelta;
	float ScaleY = deltaB/MaxDelta;
	float ScaleZ = deltaC/MaxDelta;
	
	while (1) //set output vertices and colors
	{
		if (fgets(line,150, fp) == NULL) break;
		if(strcmp(line, "\n") == 0) 
		{
			state++;
			continue;
		}
		i++;
		if (i<5) continue;
		if (state==0)
		{
			continue;
		}
		
		if (state==1)
		{
			token = strtok(line, " ");
			if(strcmp(token, "CELLS") == 0) 
			{	
				token = strtok(NULL," ");
				int hh = atoll(token);
				cells = hh;
				points = 4 * hh;
				continue;
			}
			p=0;
			while( token != NULL ) 
			{
				if (p==0) 
				{
					p++;
					token = strtok(NULL," ");
					continue;
				}
				int h = atoll(token);
				output[3*m] = remap(input[3*h],MinX,MaxX,0.0f,ScaleX) + ox;
				output[3*m+1] = remap(input[3*h+1],MinY,MaxY,0.0f,ScaleY) + oy;
				output[3*m+2] = remap(input[3*h+2],MinZ,MaxZ,0.0f,ScaleZ) + oz;
				outputColors[m] = remap(inputColors[h],CMin,CMax,ColorRangeMin,ColorRangeMax);
				m++;
				token = strtok(NULL," ");
			}
		}
		
		if (state==2)
		{
			continue;
		}

		if (state==3)
		{
			continue;
		}
	} 
	
	printf( "Bounds X range: %f %f [remapped to range 0.0 %f]\n", MinX,MaxX,ScaleX );
	printf( "Bounds Y range: %f %f [remapped to range 0.0 %f]\n", MinY,MaxY,ScaleY );
	printf( "Bounds Z range: %f %f [remapped to range 0.0 %f]\n", MinZ,MaxZ,ScaleZ );
	grid.vertices = output;
	grid.scalars = outputColors;
	return grid;
}

// Execute CUDA kernel
unsigned char* LoadCUDATexture(const char* filename, int msize, float CRMin, float CRMax, float ox, float oy, float oz)
{
	unsigned char* host = (unsigned char*) malloc(msize*msize*msize*sizeof(int)); 
	hipMalloc(&device, msize*msize*msize*sizeof(int));	
	struct DataSet k = StreamData(filename, CRMin, CRMax, ox, oy, oz);
	float* coords;
	float* colors;
	hipMalloc(&coords, 3*points*sizeof(float));
	hipMemcpy(coords, k.vertices, 3*points*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&colors, points*sizeof(float));	
	hipMemcpy(colors, k.scalars, points*sizeof(float), hipMemcpyHostToDevice);
	dim3 block(8,8,8); 
	dim3 grid(msize / block.x, msize / block.y, msize / block.z);
	printf( "%s\n", "Voxelizing scene..." );
	for (int j=0; j<cells; j++)
	{
		printf("\rPlease wait... %d percents...", (int)((float)j/(float)cells*100.0f));
		GenerateVoxelMap <<< grid, block >>>(device,coords,colors,j,msize);
	}
	hipMemcpy(host, device, msize*msize*msize*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(coords);
	hipFree(colors);
	return host;
}

unsigned char* LoadVoxelMapFromFile (const char* filename, int msize)
{
	FILE* binFile = fopen(filename,"rb");
	unsigned char* source = (unsigned char*) malloc(msize*msize*msize*sizeof(int));
	fread(source, sizeof(unsigned char), msize*msize*msize, binFile);
	fclose(binFile);
	return source;
}

void SaveVoxelMapToFile (const char* filename, unsigned char* data, int msize)
{
	FILE* binFile = fopen(filename,"wb");
	fwrite(data,sizeof(unsigned char),msize*msize*msize,binFile);
	fclose(binFile);		
}

// Set 3D volumetric texture in OpenGL environment
void SetTexture(int unit, int id, int shader, const char *name, unsigned char* source,int msize)
{
	glActiveTexture(unit);
	glBindTexture(0x806F, id);	
	glTexImage3D(0x806F,0,0x8229,msize,msize,msize,0,GL_RED,GL_UNSIGNED_BYTE,source);
	glTexParameteri(0x806F,GL_TEXTURE_MIN_FILTER,GL_LINEAR);
	glTexParameteri(0x806F,GL_TEXTURE_MAG_FILTER,GL_LINEAR);
	int loc = glGetUniformLocation(shader, name);
	glUniform1i(loc, id);
}

void MouseLook(HWND hwnd, float w, float h)
{	
	if (GetForegroundWindow()!=hwnd) return;
	POINT point;
	int mx = (int)w >> 1;
	int my = (int)h >> 1;
	GetCursorPos(&point);
	if( (point.x == mx) && (point.y == my) ) return;
	SetCursorPos(mx, my);	
	float deltaZ = (float)((mx - point.x)) ;
	float deltaX = (float)((my - point.y)) ;
	if (deltaX>0.0f) offsetX-=0.5f; 
	if (deltaX<0.0f) offsetX+=0.5f; 
	if (deltaZ>0.0f) offsetZ-=0.5f; 
	if (deltaZ<0.0f) offsetZ+=0.5f; 
	CameraRotationXMatrix[1][1] = cos(deg2rad(offsetX));
	CameraRotationXMatrix[1][2] = (-1.0f)*sin(deg2rad(offsetX));
	CameraRotationXMatrix[2][1] = sin(deg2rad(offsetX));
	CameraRotationXMatrix[2][2] = cos(deg2rad(offsetX));				
	CameraRotationYMatrix[0][0] = cos(deg2rad(offsetZ));
	CameraRotationYMatrix[0][2] = sin(deg2rad(offsetZ));
	CameraRotationYMatrix[2][0] = (-1.0f)*sin(deg2rad(offsetZ));
	CameraRotationYMatrix[2][2] = cos(deg2rad(offsetZ));
}
		
void KeyboardMovement(HWND hwnd)
{
	if (GetForegroundWindow()!=hwnd) return;
	float forward[3] = {ViewMatrix[2][0],ViewMatrix[2][1],ViewMatrix[2][2]};
	float strafe[3] = {ViewMatrix[0][0],ViewMatrix[1][0],ViewMatrix[2][0]};
	float dz = 0.0f;
	float dx = 0.0f;
	if (GetAsyncKeyState(0x57)) dz =  2.0f;
	if (GetAsyncKeyState(0x53)) dz = -2.0f ;
	if (GetAsyncKeyState(0x44)) dx =  2.0f;
	if (GetAsyncKeyState(0x41)) dx = -2.0f ;
	if (GetAsyncKeyState(0x45)) CameraTranslationMatrix[1][3] += 0.001f ;
	if (GetAsyncKeyState(0x51)) CameraTranslationMatrix[1][3] -= 0.001f ; 
	float eyeVector[3] = {CameraTranslationMatrix[0][3],CameraTranslationMatrix[1][3] ,CameraTranslationMatrix[2][3]};
	eyeVector[0] += (-dz * forward[0] + dx * strafe[0]) * 0.001f;
	eyeVector[1] += (-dz * forward[1] + dx * strafe[1]) * 0.001f;
	eyeVector[2] += (-dz * forward[2] + dx * strafe[2]) * 0.001f;
	CameraTranslationMatrix[0][3] = eyeVector[0];
	CameraTranslationMatrix[1][3] = eyeVector[1];
	CameraTranslationMatrix[2][3] = eyeVector[2];
	DebugImage = false;
	if (GetAsyncKeyState(0x58) & 0x8000) DebugImage = true;
}

// Slice volumetric texture
void Slicer (int smin, int smax)
{
	if (GetAsyncKeyState(0x55)) {sminx -= 0.001f; clamp(sminx,0.0f,1.0f);} //u
	if (GetAsyncKeyState(0x49)) {sminx += 0.001f; clamp(sminx,0.0f,1.0f);} //i
	if (GetAsyncKeyState(0x4F)) {smaxx -= 0.001f; clamp(smaxx,0.0f,1.0f);} //o
	if (GetAsyncKeyState(0x50)) {smaxx += 0.001f; clamp(smaxx,0.0f,1.0f);} //p
	if (GetAsyncKeyState(0x48)) {sminy -= 0.001f; clamp(sminy,0.0f,1.0f);} //h
	if (GetAsyncKeyState(0x4A)) {sminy += 0.001f; clamp(sminy,0.0f,1.0f);} //j
	if (GetAsyncKeyState(0x4B)) {smaxy -= 0.001f; clamp(smaxy,0.0f,1.0f);} //k
	if (GetAsyncKeyState(0x4C)) {smaxy += 0.001f; clamp(smaxy,0.0f,1.0f);} //l
	if (GetAsyncKeyState(0x56)) {sminz -= 0.001f; clamp(sminz,0.0f,1.0f);} //v
	if (GetAsyncKeyState(0x42)) {sminz += 0.001f; clamp(sminz,0.0f,1.0f);} //b
	if (GetAsyncKeyState(0x4E)) {smaxz -= 0.001f; clamp(smaxz,0.0f,1.0f);} //n
	if (GetAsyncKeyState(0x4D)) {smaxz += 0.001f; clamp(smaxz,0.0f,1.0f);} //m
	glUniform3f(smin, sminx, sminy, sminz);
	glUniform3f(smax, smaxx, smaxy, smaxz);	
}

// Diagnostic
void PrintDeviceName()
{
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	printf( "%s\n", device.name );
	printf( "CUDA compute capability: %d\n", device.major );
}

void Release ()
{
	free (input);
	free (output);
	free (inputColors);
	free (outputColors);
	hipFree(device);
}

static LRESULT CALLBACK WindowProc( HWND hWnd, UINT uMsg, WPARAM wParam, LPARAM lParam )
{
	if( uMsg==WM_SYSCOMMAND && (wParam==SC_SCREENSAVE || wParam==SC_MONITORPOWER) )
		return 0;
	if( uMsg==WM_CLOSE || uMsg==WM_DESTROY || (uMsg==WM_KEYDOWN && wParam==VK_ESCAPE) )
	{
		PostQuitMessage(0);
		return 0;
	}
	if( uMsg==WM_SIZE )
	{
		glViewport( 0, 0, lParam&65535, lParam>>16 );
	}
	if( uMsg==WM_CHAR || uMsg==WM_KEYDOWN)
	{
		if( wParam==VK_ESCAPE )
		{
			PostQuitMessage(0);
			return 0;
		}
	}
	return(DefWindowProc(hWnd,uMsg,wParam,lParam));
}

// Main function
int WINAPI WinMain( HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpCmdLine, int nCmdShow )
{
	AllocConsole();
	freopen("CONOUT$", "w+", stdout);
	int argc;
	char** argv;
	LPWSTR* lpArgv = CommandLineToArgvW( GetCommandLineW(), &argc );
	argv = (char**)malloc( argc*sizeof(char*) );
	int size = 0;
	for( int i=0; i < argc; ++i )
	{
		size = wcslen( lpArgv[i] ) + 1;
		argv[i] = (char*)malloc( size );
		wcstombs( argv[i], lpArgv[i], size );
	}
	if (argc < 21)
	{
		printf( "Not enough arguments. Exit..." );
		ExitProcess(0);
	}
	LocalFree(lpArgv);
	PrintDeviceName();
	float ScreenWidth = atof(argv[14]);
	float ScreenHeight = atof(argv[15]);
	unsigned char* Source;
	if (atoi(argv[16])==0) Source = LoadCUDATexture(argv[1],atoi(argv[2]),atof(argv[12]),atof(argv[13]),atof(argv[18]),atof(argv[19]),atof(argv[20]));
	if (atoi(argv[16])==1) Source = LoadVoxelMapFromFile(argv[17],atoi(argv[2])),atoi(argv[2]);
	if (atoi(argv[16])==2) 
	{	
		SaveVoxelMapToFile(argv[17], LoadCUDATexture(argv[1],atoi(argv[2]),atof(argv[12]),atof(argv[13]),atof(argv[18]),atof(argv[19]),atof(argv[20])), atoi(argv[2]));
		printf( "Save To file. Exit..." );
		ExitProcess(0);
	}	
	MSG msg;
	int exit = 0;
	ShowCursor(0);	
	PIXELFORMATDESCRIPTOR pfd = { 0,0,PFD_DOUBLEBUFFER };
	WNDCLASS wc;
	ZeroMemory( &wc, sizeof(WNDCLASS) );
	wc.style = CS_OWNDC|CS_HREDRAW|CS_VREDRAW;
	wc.lpfnWndProc = WindowProc;
	wc.hInstance = 0;
	wc.lpszClassName = "demo";
	wc.hbrBackground =(HBRUSH)CreateSolidBrush(0x00102030);
	RegisterClass(&wc);
	HWND hwnd = CreateWindowEx(0, wc.lpszClassName, "Demo", WS_VISIBLE|WS_OVERLAPPEDWINDOW, 0, 0, ScreenWidth, ScreenHeight, 0, 0, 0, 0);
	HDC hdc = GetDC(hwnd);
	SetPixelFormat(hdc,ChoosePixelFormat(hdc,&pfd),&pfd);
	wglMakeCurrent(hdc,wglCreateContext(hdc));
	glInit();
	wglSwapIntervalEXT (VerticalSync);
	glGenVertexArrays (1, &VertexArrayID);
	glBindVertexArray (VertexArrayID);	
	glGenBuffers(1, &VertexBuffer);
	glBindBuffer(0x8892, VertexBuffer);
	glBufferData(0x8892, sizeof(vertices), vertices, 0x88E4);	
	int PS = MakeShader(VertexShader,FragmentShader);
	SetTexture(0x84C0, 0, PS, "_MainTex", Source, atoi(argv[2]));
	glUseProgram(PS);	
	int MatrixID = glGetUniformLocation(PS,"MVP"); 
	int WorldSpaceID = glGetUniformLocation(PS,"_WorldSpaceCameraPos");
	int SliceMin = glGetUniformLocation(PS,"_SliceMin");
	int SliceMax = glGetUniformLocation(PS,"_SliceMax");
	int StepsID = glGetUniformLocation(PS,"steps");	
	int ColorMin = glGetUniformLocation(PS,"_ColorMin");
	glUniform3f(ColorMin, atof(argv[4]),atof(argv[5]),atof(argv[6]));	
	int ColorMax = glGetUniformLocation(PS,"_ColorMax");
	glUniform3f(ColorMax, atof(argv[7]),atof(argv[8]),atof(argv[9]));
	int Intensity = glGetUniformLocation(PS,"_Intensity");
	glUniform1f(Intensity, atof(argv[10]));
	int Threshold = glGetUniformLocation(PS,"_Threshold");
	glUniform1f(Threshold, atof(argv[11]));	
	ProjectionMatrix[0][0] = ((1.0f/tan(deg2rad(FieldOfView/2.0f)))/(ScreenWidth/ScreenHeight));
	ProjectionMatrix[1][1] = (1.0f/tan(deg2rad(FieldOfView/2.0f)));
	ProjectionMatrix[2][2] = (-1.0f)* (FarClip+NearClip)/(FarClip-NearClip);
	ProjectionMatrix[2][3] = (-1.0f)*(2.0f*FarClip*NearClip)/(FarClip-NearClip)	;
	while( !exit )
	{
		if (DebugImage)
		{
			glDisable (GL_BLEND);
		}
		else
		{
			glEnable (GL_BLEND); 
			glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		}
		while(PeekMessage(&msg, 0, 0, 0, PM_REMOVE) )
		{
			if( msg.message==WM_QUIT ) exit = 1;
			TranslateMessage( &msg );
			DispatchMessage( &msg );
		}
		MouseLook(hwnd,ScreenWidth,ScreenHeight);
		Slicer(SliceMin,SliceMax);
		Mul(CameraRotationYMatrix,CameraRotationXMatrix,CameraRotYX);
		Mul(CameraRotYX,CameraRotationZMatrix,CameraRotYXZ);	
		Mul(CameraTranslationMatrix,CameraRotYXZ,CameraTR);
		Mul(CameraTR,CameraScaleMatrix,CameraMatrix);
		Inverse(CameraMatrix,ViewMatrix);
		Mul(ProjectionMatrix,ViewMatrix,ProjectionViewMatrix);
		Mul(ProjectionViewMatrix,ModelMatrix,MVP);	
		float MVPT[4][4] = 
		{
			MVP[0][0], MVP[1][0], MVP[2][0], MVP[3][0],
			MVP[0][1], MVP[1][1], MVP[2][1], MVP[3][1],
			MVP[0][2], MVP[1][2], MVP[2][2], MVP[3][2],
			MVP[0][3], MVP[1][3], MVP[2][3], MVP[3][3]
		};
		glEnable(GL_DEPTH_TEST);
		glDepthFunc(GL_LESS);		
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		KeyboardMovement(hwnd);
		if (DebugImage) 
			glClearColor(0.0f, 0.0f, 1.0f, 1.0f);
		else
			glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
		glUniform1i(StepsID, atoi(argv[3]) );
		glUniformMatrix4fv(MatrixID, 1, GL_FALSE, &MVPT[0][0]);
		glUniform3f(WorldSpaceID, CameraTranslationMatrix[0][3], CameraTranslationMatrix[1][3], CameraTranslationMatrix[2][3]);
		glEnableVertexAttribArray(0);
		glBindBuffer(0x8892, VertexBuffer);
		glVertexAttribPointer(0,3, GL_FLOAT, GL_FALSE, 0,(void*)0 );
		glDrawArrays(GL_TRIANGLES, 0, 12*3);
		glDisableVertexAttribArray(0);
		wglSwapLayerBuffers(hdc, WGL_SWAP_MAIN_PLANE);
	}
	Release ();
	free(Source);
	free(argv);
	return 0;
}